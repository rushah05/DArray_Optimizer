#include "hip/hip_runtime.h"
// #pragma once
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <stdio.h>

hipblasHandle_t handle;
hipsolverHandle_t csHandle;
hipStream_t stream;
hipError_t cudaStat;
hipblasStatus_t stat;
hipsolverStatus_t statusH = HIPSOLVER_STATUS_SUCCESS;


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void cuda_init(){
    hipblasCreate(&handle);
    hipsolverDnCreate(&csHandle);
    hipStreamCreate(&stream);
    hipblasSetStream(handle, stream);
    hipsolverSetStream(csHandle, stream);
}

void cuda_finalize(){
}


template<typename T>
void printMatrixDeviceBlock(char *filename,int m, int n, T* dA, int lda)
{
    FILE *f = fopen(filename, "w");
	if (f == NULL) {
		printf("fault!\n");
		return;
	}
    float *ha;
    ha = (float*)malloc(sizeof(float));

    for(int i = 0;i<m;i++){
        for(int j = 0;j<n;j++){
            hipMemcpy(&ha[0], &dA[i+j*lda], sizeof(float), hipMemcpyDeviceToHost);
            fprintf(f, "%lf", ha[0]);
            if (j == n - 1) fprintf(f, "\n");
			else fprintf(f, " ,");
        }
    }
    fclose(f);
    free(ha);
}


template<typename T>
void printVectorDeviceBlock(char *filename, int m, T* dA)
{
    FILE *f = fopen(filename, "w");
	if (f == NULL) {
		printf("fault!\n");
		return;
	}
    float *ha;
    ha = (float*)malloc(sizeof(float));

    for(int i=0; i<m; i++){
        hipMemcpy(&ha[0], &dA[i], sizeof(float), hipMemcpyDeviceToHost);
        fprintf(f, "%d", i);
        fprintf(f, "::");
        fprintf(f, "%lf", ha[0]);
        fprintf(f, "\n");
    }

    fclose(f);
    free(ha);
}


template<typename FloatType, typename GammaType>
__global__
void rbf( int m, int n, FloatType *buf, int ldb, FloatType *Xi, FloatType *Xj, FloatType *Xij, int ldxij, GammaType gamma, FloatType *Yi, FloatType *Yj)
{
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	int j=blockIdx.y*blockDim.y + threadIdx.y;

	if (i<m && j<n) {
		buf[i+j*ldb] = Yi[i]*Yj[j]*__expf(-gamma*(Xi[i] + Xj[j] - 2*Xij[i+j*ldxij]));
		// printf("[i,j]=[%d,%d], buf[]=%.4f, XI[]=%.4f, XJ[]=%.4f, XIJ[]=%.4f\n", i, j, buf[i+j*ldb],
		// 	   XI[i], XJ[j], XIJ[i+j*ldxij]);
	}
}


template<typename FloatType>
__global__ 
void vecnorm(FloatType *Zd, int ldz, FloatType *ZI, int m, int k){
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	if( i<m ) {
		FloatType sum = 0.0f;

        #pragma unroll (4)
		for( int j=0; j<k; j++ )
			sum += Zd[i+j*ldz]*Zd[i+j*ldz];

		ZI[i] = sum;
	}
}

__global__ 
void fnorm( int m, int n, double *buf, int B, double *XI, double *XJ, double *XIJ, int ldxij, double gamma, double *YI, double *YJ, double *acc){
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	int j=blockIdx.y*blockDim.y + threadIdx.y;

	if (i<m && j<n) 
		// buf[i+j*ldb] = YI[i]*YJ[j]*__expf(-gamma*(XI[i] + XJ[j] - 2*XIJ[i+j*ldxij]));
		acc += 0;
}


__global__ void s2h(float *Z, __half *hZ, int ldz, int m, int n){
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    int j=blockIdx.y*blockDim.y+threadIdx.y;

    if(i < m && j < n){
        hZ[i+j*ldz] = __float2half(Z[i+j*ldz]);
    }
}

__global__ void h2s(__half *hZ, float *Z, int ldz, int m, int n){
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    int j=blockIdx.y*blockDim.y+threadIdx.y;

    if(i < m && j < n){
        Z[i+j*ldz] = __half2float(hZ[i+j*ldz]);
    }
}


__global__ void s2h_vec(float *Z, __half *hZ, int m){
    int i=blockIdx.x*blockDim.x+threadIdx.x;

    if(i < m){
        hZ[i] = __float2half(Z[i]);
    }
}




void lra(int rank, int gn, int ln, int d, int k, float* Xi, int ldxi, float* Xj, int ldxj, float* Yi, float* Yj, float* KO, int ldk, float gamma, float* Omega, int ldo){
    const int B=16384;
    if(rank ==0) printf("gn::%d, ln::%d, d::%d, k::%d\nldxi::%d, ldxj::%d, ldk::%d, ldo::%d\n", gn, ln, d, k, ldxi, ldxj, ldk, ldo);
    float *dXi, *dXj, *dYi, *dYj, *dO, *dKO;
    gpuErrchk(hipMalloc(&dXi, sizeof(float)*ln*d));
    gpuErrchk(hipMalloc(&dXj, sizeof(float)*gn*d));
    gpuErrchk(hipMalloc(&dYi, sizeof(float)*ln));
    gpuErrchk(hipMalloc(&dYj, sizeof(float)*gn));
    gpuErrchk(hipMalloc(&dO, sizeof(float)*gn*k));
    gpuErrchk(hipMalloc(&dKO, sizeof(float)*ln*k));
    gpuErrchk(hipMemcpy(dXi, Xi, sizeof(float)*ln*d, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dXj, Xj, sizeof(float)*gn*d, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dYi, Yi, sizeof(float)*ln, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dYj, Yj, sizeof(float)*gn, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dO, Omega, sizeof(float)*gn*k, hipMemcpyHostToDevice));

    float *dXi_sqr, *dXj_sqr, *dk, *dXij;
    gpuErrchk(hipMalloc(&dXi_sqr, sizeof(float)*B));
    gpuErrchk(hipMalloc(&dXj_sqr, sizeof(float)*B));
    gpuErrchk(hipMalloc(&dk, sizeof(float)*B*B));
    gpuErrchk(hipMalloc(&dXij, sizeof(float)*B*B));
    hipDeviceSynchronize();

    float sone=1.0f;
    float szero=0.0f;
    for (int i=0; i<ln; i+=B) {
		int ib = min(B, ln-i);
		for (int j=0; j<gn; j+=B) {
			int jb = min(B, gn-j);

			// printf("[i=%d j=%d] ib::%d, jb::%d\n", i,j, ib, jb);
			// step 1: populate XI, XJ, XIJ
			vecnorm<<<(B+63)/64, 64>>>(&dXi[i], ldxi, dXi_sqr, ib, d);
            gpuErrchk( hipPeekAtLastError() );
			vecnorm<<<(B+63)/64, 64>>>(&dXj[j], ldxj, dXj_sqr, jb, d);
			gpuErrchk( hipPeekAtLastError() );

			// XIJ is column major!!
			// printf("ib=%d jb=%d d=%d ldz=%d\n", ib, jb, d, ldz);
			stat = hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, ib, jb, d,
								   &sone, &dXi[i], HIP_R_32F,ldxi,
								   &dXj[j], HIP_R_32F,ldxj, &szero,
								   dXij, HIP_R_32F, ib,
								   HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);	
			if (stat != HIPBLAS_STATUS_SUCCESS) printf ("hipblasGemmEx failed %s\n", __LINE__);
			gpuErrchk( hipPeekAtLastError() );

			dim3 threadsPerBlock(32,32);
			dim3 numBlocks((ib+threadsPerBlock.x-1)/threadsPerBlock.x,
							(jb+threadsPerBlock.y-1)/threadsPerBlock.y );

			// printf("ib=%d, jb=%d, B=%d, TPB.(x,y)=(%d,%d), B.(x,y)=(%d,%d)\n",
			// 	   ib, jb, B, threadsPerBlock.x, threadsPerBlock.y,
			// 	   numBlocks.x, numBlocks.y);
			rbf<<<numBlocks, threadsPerBlock>>>( ib, jb, dk, ib, dXi_sqr, dXj_sqr, dXij, ib, gamma, &dYi[i], &dYj[j]);
			gpuErrchk( hipPeekAtLastError());
			gpuErrchk( hipDeviceSynchronize());

            stat = hipblasGemmEx( handle, HIPBLAS_OP_N, HIPBLAS_OP_N, ib, k, jb, 
                                &sone, dk, HIP_R_32F, B, &dO[j], HIP_R_32F, ldo,
                                &sone, &dKO[i], HIP_R_32F, ldk, 
                                HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
            if (stat != HIPBLAS_STATUS_SUCCESS) printf ("hipblasGemmEx failed %s\n", __LINE__);
        }
    }

    hipDeviceSynchronize();
    gpuErrchk(hipMemcpy(KO, dKO, sizeof(float)*ln*k, hipMemcpyDeviceToHost));
    hipPeekAtLastError();
}



void SGEQRF(int *m, int *n, float *Q, int *ldq, float *tau, float *work, int *lwork, int *info){

}
