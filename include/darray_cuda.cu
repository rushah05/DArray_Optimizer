#include "hip/hip_runtime.h"
// #pragma once
#include <cassert>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <stdio.h>

hipblasHandle_t handle;
hipsolverHandle_t csHandle;
hipStream_t stream;
hipError_t cudaStat;
hipblasStatus_t stat;
hipsolverStatus_t statusH = HIPSOLVER_STATUS_SUCCESS;


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void cuda_init(){
    hipblasCreate(&handle);
    hipsolverDnCreate(&csHandle);
    hipStreamCreate(&stream);
    hipblasSetStream(handle, stream);
    hipsolverSetStream(csHandle, stream);
}

void cuda_finalize(){
}


template<typename T>
void printMatrixDeviceBlock(char *filename,int m, int n, T* dA, int lda)
{
    FILE *f = fopen(filename, "w");
	if (f == NULL) {
		printf("fault!\n");
		return;
	}
    float *ha;
    ha = (float*)malloc(sizeof(float));

    for(int i = 0;i<m;i++){
        for(int j = 0;j<n;j++){
            hipMemcpy(&ha[0], &dA[i+j*lda], sizeof(float), hipMemcpyDeviceToHost);
            fprintf(f, "%lf", ha[0]);
            if (j == n - 1) fprintf(f, "\n");
			else fprintf(f, " ,");
        }
    }
    fclose(f);
    free(ha);
}

__global__
void  getR(int m, int n, float *da, int lda, float *dr, int ldr)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	if (i < m&&j < n){
		if (i <= j){
			dr[i + j*ldr] = da[i + j*lda];
		}
	}
}



template<typename T>
void printVectorDeviceBlock(char *filename, int m, T* dA)
{
    FILE *f = fopen(filename, "w");
	if (f == NULL) {
		printf("fault!\n");
		return;
	}
    float *ha;
    ha = (float*)malloc(sizeof(float));

    for(int i=0; i<m; i++){
        hipMemcpy(&ha[0], &dA[i], sizeof(float), hipMemcpyDeviceToHost);
        fprintf(f, "%d", i);
        fprintf(f, "::");
        fprintf(f, "%lf", ha[0]);
        fprintf(f, "\n");
    }

    fclose(f);
    free(ha);
}


template<typename FloatType, typename GammaType>
__global__
void rbf( int m, int n, FloatType *buf, int ldb,
				 FloatType *XI, FloatType *XJ, FloatType *XIJ, int ldxij,
				 GammaType gamma, FloatType *YI, FloatType *YJ)
{
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	int j=blockIdx.y*blockDim.y + threadIdx.y;

	if (i<m && j<n) {
		buf[i+j*ldb] = YI[i]*YJ[j]*__expf(-gamma*(XI[i] + XJ[j] - 2*XIJ[i+j*ldxij]));
		// printf("[m,n]=[%d,%d], [i,j]=[%d,%d], buf[]=%.4f, XI[]=%.4f, XJ[]=%.4f, XIJ[]=%.4f\n", m, n, i, j, buf[i+j*ldb],
		// 	   XI[i], XJ[j], XIJ[i+j*ldxij]);
	}
}



template<typename FloatType>
__global__ 
void vecnorm(FloatType *Zd, int ldz, FloatType *ZI, int m, int k){
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	if( i<m ) {
		FloatType sum = 0.0f;

        #pragma unroll (4)
		for( int j=0; j<k; j++ )
			sum += Zd[i+j*ldz]*Zd[i+j*ldz];

		ZI[i] = sum;
	}
}

__global__ 
void fnorm( int m, int n, double *buf, int B, double *XI, double *XJ, double *XIJ, int ldxij, double gamma, double *YI, double *YJ, double *acc){
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	int j=blockIdx.y*blockDim.y + threadIdx.y;

	if (i<m && j<n) 
		// buf[i+j*ldb] = YI[i]*YJ[j]*__expf(-gamma*(XI[i] + XJ[j] - 2*XIJ[i+j*ldxij]));
		acc += 0;
}


__global__ void s2h(float *Z, __half *hZ, int ldz, int m, int n){
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    int j=blockIdx.y*blockDim.y+threadIdx.y;

    if(i < m && j < n){
        hZ[i+j*ldz] = __float2half(Z[i+j*ldz]);
    }
}

__global__ void h2s(__half *hZ, float *Z, int ldz, int m, int n){
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    int j=blockIdx.y*blockDim.y+threadIdx.y;

    if(i < m && j < n){
        Z[i+j*ldz] = __half2float(hZ[i+j*ldz]);
    }
}

void transpose(float *dA, int lda, int m, int n)
{
	float *dC;
	gpuErrchk(hipMalloc(&dC, sizeof(float)*m*n));
	float alpha = 1.0;
	float beta = 0.0;
	hipblasSgeam(handle,
		HIPBLAS_OP_T, HIPBLAS_OP_T,
		m, n,
		&alpha,
		dA, lda,
		&beta,
		dA, lda,
		dC, m);
	hipMemcpyAsync(dA, dC, m * n * sizeof(float), hipMemcpyDeviceToDevice, stream);
	hipFree(dC);
}

__global__ void gemm(float *dA, int lda, float *dB, int ldb, float *dC, int ldc, int m, int k, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if(i < m && j < n) {
        float sum = 0.0f;
        for(int d=0; d<k; ++d){
            sum += dA[i+d*lda] * dB[d+j*ldb];
        }
        dC[i+j*ldc] = sum;
    }
}

void LRA(int rank, int lm, int ln, int ld, int lk, float* Xi, int ldxi, float* Xj, int ldxj, float* Yi, float* Yj, float* A, int lda, float gamma, float* O, int ldo){
    // printf("[Rank::%d] lm::%d, ln::%d, ld::%d, lk::%d \nldxi::%d, ldxj::%d, lda::%d, ldo::%d\n", rank, lm, ln, ld, lk, ldxi, ldxj, lda, ldo);
    const int B=8192;
    float *dXi, *dXj, *dXij, *dYi, *dYj, *dk, *dO, *dA;
    gpuErrchk(hipMalloc(&dXi, sizeof(float)*lm*ld));
    gpuErrchk(hipMalloc(&dXj, sizeof(float)*ln*ld));
    gpuErrchk(hipMalloc(&dYi, sizeof(float)*lm));
    gpuErrchk(hipMalloc(&dYj, sizeof(float)*ln));
    gpuErrchk(hipMalloc(&dXij, sizeof(float)*B*B));
    gpuErrchk(hipMalloc(&dk, sizeof(float)*B*B));
    gpuErrchk(hipMalloc(&dO, sizeof(float)*ln*lk));
    gpuErrchk(hipMalloc(&dA, sizeof(float)*lm*lk));
    gpuErrchk(hipMemset(dA, 0.0, lm*lk));
    gpuErrchk(hipMemcpy(dXi, Xi, sizeof(float)*lm*ld, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dXj, Xj, sizeof(float)*ln*ld, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dYi, Yi, sizeof(float)*lm, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dYj, Yj, sizeof(float)*ln, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dO, O, sizeof(float)*ln*lk, hipMemcpyHostToDevice));
    
    float *dXi_sqr, *dXj_sqr;
    gpuErrchk(hipMalloc(&dXi_sqr, sizeof(float)*B));
    gpuErrchk(hipMalloc(&dXj_sqr, sizeof(float)*B));
    float sone=1.0f;
    float szero=0.0f;
    dim3 threadsPerBlock(32,32);
   

    for(int i=0; i<lm; i+=B){
        int ib=min(B, lm-i);
        for(int j=0; j<ln; j+=B){
            int jb=min(B, ln-j);
            vecnorm<<<(ib+63)/64, 64>>>(&dXi[i], ldxi, dXi_sqr, ib, ld);
	        vecnorm<<<(jb+63)/64, 64>>>(&dXj[j], ldxj, dXj_sqr, jb, ld);
            stat = hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, ib, jb, ld, &sone, &dXi[i], HIP_R_32F, ldxi,
                            &dXj[j], HIP_R_32F, ldxj, &szero, dXij, HIP_R_32F, ib,
                            HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);	
            if (stat != HIPBLAS_STATUS_SUCCESS) printf ("hipblasGemmEx failed %s\n", __LINE__);
            // printMatrixDeviceBlock("Xij.csv", ib, jb, dXij, B);
            // printVectorDeviceBlock("Xi_sqr.csv", ib, dXi_sqr);
            // printVectorDeviceBlock("Xj_sqr.csv", jb, dXj_sqr);
            // printVectorDeviceBlock("Yi.csv", ib, &dYi[i]);
            // printVectorDeviceBlock("Yj.csv", jb, &dYj[j]);

            dim3 numBlocks((ib+threadsPerBlock.x-1)/threadsPerBlock.x, (jb+threadsPerBlock.y-1)/threadsPerBlock.y );
            rbf<<<numBlocks, threadsPerBlock>>>(ib, jb, dk, ib, dXi_sqr, dXj_sqr, dXij, _IOFBF, gamma, &dYi[i], &dYj[j]);

            // printMatrixDeviceBlock("K.csv", ib, jb, dk, ib);
            // printMatrixDeviceBlock("O.csv", jb, lk, &dO[j], ldo);
             stat = hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, ib, lk, jb, &sone, dk, HIP_R_32F, ib,
                            &dO[j], HIP_R_32F, ldo, &szero, &dA[i], HIP_R_32F, lda,
                            HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);	
            if (stat != HIPBLAS_STATUS_SUCCESS) printf ("hipblasGemmEx failed %s\n", __LINE__);
        }
    }

    // if(rank == 0) printMatrixDeviceBlock("A0.csv", lm, lk, dA, lda);
    gpuErrchk(hipMemcpy(A, dA, sizeof(float)*lm*lk, hipMemcpyDeviceToHost));
    hipDeviceSynchronize();
    hipFree(dXi);
    hipFree(dXj);
    hipFree(dXij);
    hipFree(dk);
    hipFree(dO);
    hipFree(dA);
    hipFree(dXi_sqr);
    hipFree(dXj_sqr);
}


// void LRA(int rank, int lm, int ln, int ld, int lk, float* Xi, int ldxi, float* Xj, int ldxj, float* Yi, float* Yj, float* A, int lda, float gamma, float* O, int ldo){
//     // printf("[Rank::%d] lm::%d, ln::%d, ld::%d, lk::%d \nldxi::%d, ldxj::%d, lda::%d, ldo::%d\n", rank, lm, ln, ld, lk, ldxi, ldxj, lda, ldo);
//     const int B=8192;
//     float *dXi, *dXj, *dXij, *dYi, *dYj, *dk, *dO, *dA;
//     gpuErrchk(hipMalloc(&dXi, sizeof(float)*lm*ld));
//     gpuErrchk(hipMalloc(&dXj, sizeof(float)*ln*ld));
//     gpuErrchk(hipMalloc(&dYi, sizeof(float)*lm));
//     gpuErrchk(hipMalloc(&dYj, sizeof(float)*ln));
//     gpuErrchk(hipMalloc(&dXij, sizeof(float)*B*B));
//     gpuErrchk(hipMalloc(&dk, sizeof(float)*B*B));
//     gpuErrchk(hipMalloc(&dO, sizeof(float)*ln*lk));
//     gpuErrchk(hipMalloc(&dA, sizeof(float)*lm*lk));
//     gpuErrchk(hipMemset(dA, 0.0, lm*lk));
//     gpuErrchk(hipMemcpy(dXi, Xi, sizeof(float)*lm*ld, hipMemcpyHostToDevice));
//     gpuErrchk(hipMemcpy(dXj, Xj, sizeof(float)*ln*ld, hipMemcpyHostToDevice));
//     gpuErrchk(hipMemcpy(dYi, Yi, sizeof(float)*lm, hipMemcpyHostToDevice));
//     gpuErrchk(hipMemcpy(dYj, Yj, sizeof(float)*ln, hipMemcpyHostToDevice));
//     gpuErrchk(hipMemcpy(dO, O, sizeof(float)*ln*lk, hipMemcpyHostToDevice));
    
//     float *dXi_sqr, *dXj_sqr;
//     gpuErrchk(hipMalloc(&dXi_sqr, sizeof(float)*B));
//     gpuErrchk(hipMalloc(&dXj_sqr, sizeof(float)*B));
//     float sone=1.0f;
//     float szero=0.0f;
//     dim3 threadsPerBlock(32,32);
   

//     for(int i=0; i<lm; i+=B){
//         int ib=min(B, lm-i);
//         for(int j=0; j<ln; j+=B){
//             int jb=min(B, ln-j);
//             vecnorm<<<(ib+63)/64, 64>>>(dXi, ldxi, dXi_sqr, ib, ld);
// 	        vecnorm<<<(jb+63)/64, 64>>>(dXj, ldxj, dXj_sqr, jb, ld);
//             stat = hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, ib, jb, ld, &sone, &dXi[i], HIP_R_32F, ldxi,
//                             &dXj[j], HIP_R_32F, ldxj, &szero, dXij, HIP_R_32F, ib,
//                             HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);	
//             if (stat != HIPBLAS_STATUS_SUCCESS) printf ("hipblasGemmEx failed %s\n", __LINE__);
//             // printMatrixDeviceBlock("Xij.csv", ib, jb, dXij, B);
//             // printVectorDeviceBlock("Xi_sqr.csv", ib, dXi_sqr);
//             // printVectorDeviceBlock("Xj_sqr.csv", jb, dXj_sqr);
//             // printVectorDeviceBlock("Yi.csv", ib, &dYi[i]);
//             // printVectorDeviceBlock("Yj.csv", jb, &dYj[j]);

//             dim3 numBlocks((ib+threadsPerBlock.x-1)/threadsPerBlock.x, (jb+threadsPerBlock.y-1)/threadsPerBlock.y );
//             rbf<<<numBlocks, threadsPerBlock>>>(ib, jb, dk, ib, dXi_sqr, dXj_sqr, dXij, _IOFBF, gamma, &dYi[i], &dYj[j]);

//             // printMatrixDeviceBlock("K.csv", ib, jb, dk, ib);
//             // printMatrixDeviceBlock("O.csv", jb, lk, &dO[j], ldo);
//              stat = hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, ib, lk, jb, &sone, dk, HIP_R_32F, ib,
//                             &dO[j], HIP_R_32F, ldo, &szero, &dA[i], HIP_R_32F, lda,
//                             HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);	
//             if (stat != HIPBLAS_STATUS_SUCCESS) printf ("hipblasGemmEx failed %s\n", __LINE__);
//         }
//     }

//     // if(rank == 0) printMatrixDeviceBlock("A0.csv", lm, lk, dA, lda);
//     gpuErrchk(hipMemcpy(A, dA, sizeof(float)*lm*lk, hipMemcpyDeviceToHost));
//     hipDeviceSynchronize();
//     hipFree(dXi);
//     hipFree(dXj);
//     hipFree(dXij);
//     hipFree(dk);
//     hipFree(dO);
//     hipFree(dA);
//     hipFree(dXi_sqr);
//     hipFree(dXj_sqr);
// }


void SGEQRF(int m, int n, float *Q, int ldq){
    float *dQ, *d_tau, *d_work;
    int *d_info, info;
    int lwork_geqrf = 0;
	int lwork_orgqr = 0;
    int lwork = 0;

    gpuErrchk(hipMalloc(&dQ, sizeof(float)*m*n));
    gpuErrchk(hipMalloc(&d_tau, sizeof(float)*n));
	gpuErrchk(hipMalloc((void**)&d_info, sizeof(int)));
    gpuErrchk(hipMemcpy(dQ, Q, sizeof(float)*m*n, hipMemcpyHostToDevice));

    statusH = hipsolverDnSgeqrf_bufferSize(csHandle, m, n, dQ, ldq, &lwork_geqrf);
    assert(statusH == HIPSOLVER_STATUS_SUCCESS);
    statusH = hipsolverDnSorgqr_bufferSize(csHandle, m, n, n, dQ, ldq, d_tau, &lwork_orgqr);
    assert(statusH == HIPSOLVER_STATUS_SUCCESS);
    // printf("lwork_geqrf::%d, lwork_orgqr::%d\n", lwork_geqrf, lwork_orgqr);

	lwork = (lwork_geqrf > lwork_orgqr) ? lwork_geqrf : lwork_orgqr;
	hipMalloc(&d_work, sizeof(int)*lwork);

    statusH = hipsolverDnSgeqrf(csHandle, m, n, dQ, ldq, d_tau, d_work, lwork, d_info);
    assert(statusH == HIPSOLVER_STATUS_SUCCESS);
    statusH = hipsolverDnSorgqr(csHandle, m, n, n, dQ, ldq, d_tau, d_work, lwork, d_info);
    assert(statusH == HIPSOLVER_STATUS_SUCCESS);
    gpuErrchk(hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost));
    if (info!=0) {
        printf("cusolver Sorgqr fail; info=%d\n", info);
        exit(1); 
    }
       
    gpuErrchk(hipMemcpy(Q, dQ, sizeof(float)*m*n, hipMemcpyDeviceToHost));

    hipFree(dQ);
    hipFree(d_tau);
    hipFree(d_work);
    hipFree(d_info);
}

void SORMQR(int m, int n, float *Q, int ldq, float *RQ, int ldrq){
    float *dQ, *dRQ, *d_tau, *d_work;
    int *d_info, info;
	int lwork_ormqr=0;

    gpuErrchk(hipMalloc(&dQ, sizeof(float)*m*n));
    gpuErrchk(hipMalloc(&dRQ, sizeof(float)*m*n));
    gpuErrchk(hipMalloc(&d_tau, sizeof(float)*m));
	gpuErrchk(hipMalloc((void**)&d_info, sizeof(int)));
    gpuErrchk(hipMemcpy(dQ, Q, sizeof(float)*m*n, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dRQ, RQ, sizeof(float)*n*n, hipMemcpyHostToDevice));

    statusH = hipsolverDnSormqr_bufferSize(csHandle, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_N, m, n, n, dQ, ldq, d_tau, dRQ, ldrq, &lwork_ormqr);
	assert(statusH == HIPSOLVER_STATUS_SUCCESS);
    hipMalloc(&d_work, sizeof(int)*lwork_ormqr);

    hipsolverDnSormqr(csHandle, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_N, m, n, n, dQ, ldq, d_tau, dRQ, ldrq, d_work, lwork_ormqr, d_info);
    gpuErrchk(hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost));
    if (info!=0) {
        printf("cusolver Sormqr fail; info=%d\n", info);
        exit(1); 
    }

    gpuErrchk(hipMemcpy(RQ, dRQ, sizeof(float)*m*n, hipMemcpyDeviceToHost));
    hipFree(dQ);
    hipFree(dRQ);
    hipFree(d_tau);
    hipFree(d_work);
    hipFree(d_info);
}


void TCGemm(int rank, int m, int n, int k, float* A, int lda, float* B, int ldb, float alpha, float beta, float* C, int ldc){
    // printf("[Rank::%d] m::%d, n::%d, k::%d lda::%d, ldb::%d, ldc::%d\n", rank, m, n, k, lda, ldb, ldc);
    float *da, *db, *dc;
    gpuErrchk(hipMalloc(&da, sizeof(float)*m*k));
    gpuErrchk(hipMalloc(&db, sizeof(float)*k*n));
    gpuErrchk(hipMalloc(&dc, sizeof(float)*m*n));
    gpuErrchk(hipMemcpy(da, A, sizeof(float)*m*k, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(db, B, sizeof(float)*k*n, hipMemcpyHostToDevice));

    stat = hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, da, HIP_R_32F, lda,
                        db, HIP_R_32F, ldb, &beta, dc, HIP_R_32F, ldc, HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);	
    if (stat != HIPBLAS_STATUS_SUCCESS) printf ("hipblasGemmEx failed %s\n", __LINE__);
    gpuErrchk(hipMemcpy(C, dc, sizeof(float)*m*n, hipMemcpyDeviceToHost));

    hipDeviceSynchronize();
    hipFree(da);
    hipFree(db);
    hipFree(dc); 
}

void Chol(int k, float *C, int ldc){
    float *Cd, *d_work;
    gpuErrchk(hipMalloc(&Cd, sizeof(float)*k*k));
    gpuErrchk(hipMemcpy(Cd, C, sizeof(float)*k*k, hipMemcpyHostToDevice));

    int info, *d_info;
    int lwork=0; 
    gpuErrchk(hipMalloc(&d_info, sizeof(int)));
    statusH = hipsolverDnSpotrf_bufferSize(csHandle, HIPBLAS_FILL_MODE_LOWER, k, Cd, ldc, &lwork);
    assert(statusH == HIPSOLVER_STATUS_SUCCESS);

    gpuErrchk(hipMalloc(&d_work, sizeof(float)*lwork));
    statusH = hipsolverDnSpotrf(csHandle, HIPBLAS_FILL_MODE_LOWER, k, Cd, ldc, d_work, lwork, d_info);
    assert(statusH == HIPSOLVER_STATUS_SUCCESS);
    hipFree(d_work);
    gpuErrchk(hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost) );
    if (info!=0) {
        printf("Cholesky fail; info=%d\n", info);
        // exit(1); 
    }
    hipFree(d_info); 
}













