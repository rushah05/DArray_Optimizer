#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>

#include <stdio.h>

/*
cd ../tests
module load GCC/7.3.0
nvcc -c test_cuda.cu -o cuda.o
cd ../build
make
module load GCC
./test_lra
*/

float *mem;
__half *half_mem;
float *dA, *dB, *dC;
__half *halfA, *halfB;
hipblasHandle_t handle;

template<typename T>
void printMatrixDeviceBlock(char *filename,int m, int n, T* dA, int lda)
{
    FILE *f = fopen(filename, "w");
	if (f == NULL) {
		printf("fault!\n");
		return;
	}
    //printf("Perform printmatrixdevice\n");
    float *ha;
    ha = (float*)malloc(sizeof(float));

    for(int i = 0;i<m;i++)
    {
        for(int j = 0;j<n;j++)
        {
            hipMemcpy(&ha[0], &dA[i+j*lda], sizeof(float), hipMemcpyDeviceToHost);
            fprintf(f, "%lf", ha[0]);
            if (j == n - 1) fprintf(f, "\n");
			else fprintf(f, ",");
        }
    }
    fclose(f);
	//cudaMemcpy(ha, dA, sizeof(float)*m*n, cudaMemcpyDeviceToHost);
    //printMatrixFloat(filename, m, n, ha, lda);
    free(ha);
}

void init()
{
	hipMalloc(&mem, sizeof(float)*65536*65536);
	hipMalloc(&half_mem, sizeof(__half)*65536*65536);
	hipblasCreate(&handle);
}

void finalize()
{
	hipFree(mem);
	hipFree(half_mem);
}

void host2device(int m, int n, int k, float *hA, float *hB)
{
	dA = mem;
	dB = mem+sizeof(float)*m*k;
	dC = mem+sizeof(float)*k*n+sizeof(float)*m*k;
	hipMemcpy(dA, hA, sizeof(float)*m*k, hipMemcpyHostToDevice);
	hipMemcpy(dB, hB, sizeof(float)*k*n, hipMemcpyHostToDevice);
	halfA = half_mem;
	halfB = half_mem+sizeof(__half)*m*k;
}

void device2host(int m, int n, float *hC)
{
	hipMemcpy(hC, dC, sizeof(float)*m*n, hipMemcpyDeviceToHost);
}


__global__
void s2h(int m, int n, float *as, int ldas, __half *ah, int ldah)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	if (i < m && j < n) {
		ah[i + j*ldah] = __float2half(as[i + j*ldas]);
	}
}

void transpose(float *dA, int lda, int m, int n)
{
    hipblasHandle_t handle;
    hipblasCreate(&handle);
	float *dC;
	hipMalloc(&dC, sizeof(float)*m*n);
	float alpha = 1.0;
	float beta = 0.0;
	hipblasSgeam(handle,
		HIPBLAS_OP_T, HIPBLAS_OP_T,
		m, n,
		&alpha,
		dA, lda,
		&beta,
		dA, lda,
		dC, lda);
	hipMemcpy(dA, dC, m * n * sizeof(float), hipMemcpyDeviceToDevice);
	hipFree(dC);
}

void TCGEMM(int m, int n, int k, float alpha, float beta, float *hA, float *hB, float *hC)
{
	printf("m,n,k=%d,%d,%d\n", m, n, k);
	host2device(m,n,k,hA,hB);
	//printMatrixDeviceBlock("dA.csv",m, k, dA, m);
	//printMatrixDeviceBlock("dB.csv",k, n, dB, n);
	dim3 gridDimA((m+31)/32, (k+31)/31 );
	dim3 blockDim(32,32);
	s2h<<<gridDimA, blockDim>>>(m,k, dA, m, halfA, m);
	dim3 gridDimB((k+31)/32, (n+31)/31);
	s2h<<<gridDimB, blockDim>>>(k,n, dB, k, halfB, k);

	hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k,
        &alpha, halfA, HIP_R_16F, m, halfB, HIP_R_16F, k,
        &beta, dC, HIP_R_32F, m, HIP_R_32F,
        CUBLAS_GEMM_DEFAULT_TENSOR_OP
    );

	//transpose(dC, m, m, n);

	device2host(m,n,hC);
}

void test_cuda()
{
	printf("Yes\n");
}
